#include "hip/hip_runtime.h"
/**
* This file is part of RGBID-SLAM.
*
* Copyright (C) 2015 Daniel Gutiérrez Gómez <danielgg at unizar dot es> (Universidad de Zaragoza)
*
* RGBID-SLAM is free software: you can redistribute it and/or modify
* it under the terms of the GNU General Public License as published by
* the Free Software Foundation, either version 3 of the License, or
* (at your option) any later version.
*
* RGBID-SLAM is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
* GNU General Public License for more details.
*
* You should have received a copy of the GNU General Public License
* along with RGBID-SLAM. If not, see <http://www.gnu.org/licenses/>.
*/

/**
 * Software License Agreement (BSD License)
 *
 *  Point Cloud Library (PCL) - www.pointclouds.org
 *  Copyright (c) 2011, Willow Garage, Inc.
 *
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of Willow Garage, Inc. nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include "device.hpp"

namespace RGBID_SLAM
{
  namespace device
	{
    struct ImageGenerator
    {
      enum
      {
        CTA_SIZE_X = 32, CTA_SIZE_Y = 8
      };

      PtrStep<float> vmap;
      PtrStep<float> nmap;

      LightSource light;

      mutable PtrStepSz<uchar3> dst;

      __device__ __forceinline__ void
      operator () () const
      {
        int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
        int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

        if (x >= dst.cols || y >= dst.rows)
          return;

        float3 v, n;
        v.x = vmap.ptr (y)[x];
        n.x = nmap.ptr (y)[x];

        uchar3 color = make_uchar3 (0, 0, 0);

        if (!isnan (v.x) && !isnan (n.x))
        {
          v.y = vmap.ptr (y + dst.rows)[x];
          v.z = vmap.ptr (y + 2 * dst.rows)[x];

          n.y = nmap.ptr (y + dst.rows)[x];
          n.z = nmap.ptr (y + 2 * dst.rows)[x];

          float weight = 1.f;

          for (int i = 0; i < light.number; ++i)
          {
            float3 vec = normalized (light.pos[i] - v);

            weight *= fabs (dot (vec, n));
          }

          int br = (int)(205 * weight) + 50;
          br = max (0, min (255, br));
          color = make_uchar3 (br, br, br);
        }
        dst.ptr (y)[x] = color;
      }
    };

    __global__ void
    generateImageKernel (const ImageGenerator ig) {
      ig ();
    }
    
    struct ImageGeneratorRGB
    {
      enum
      {
        CTA_SIZE_X = 32, CTA_SIZE_Y = 8
      };

      PtrStep<float> vmap;
      PtrStep<float> nmap;
      PtrStep<uchar3> rgb;
      

      LightSource light;

      mutable PtrStepSz<uchar3> dst;

      __device__ __forceinline__ void
      operator () () const
      {
        int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
        int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

        if (x >= dst.cols || y >= dst.rows)
          return;

        float3 v, n;
        v.x = vmap.ptr (y)[x];
        n.x = nmap.ptr (y)[x];

        uchar3 color = make_uchar3 (0, 0, 0);

        if (!isnan (v.x) && !isnan (n.x))
        {
          v.y = vmap.ptr (y + dst.rows)[x];
          v.z = vmap.ptr (y + 2 * dst.rows)[x];

          n.y = nmap.ptr (y + dst.rows)[x];
          n.z = nmap.ptr (y + 2 * dst.rows)[x];

          float weight = 1.f;

          for (int i = 0; i < light.number; ++i)
          {
            float3 vec = normalized (light.pos[i] - v);

            weight *= fabs (dot (vec, n));
          }

          int br = (int)(205 * weight) + 50;
          br = max (0, min (255, br));
          float br_f = __int2float_rn(br) / 255.f;
          int r = __float2int_rn(__int2float_rn((rgb.ptr (y)[x]).x) * br_f );
          int g = __float2int_rn(__int2float_rn((rgb.ptr (y)[x]).y) * br_f );
          int b = __float2int_rn(__int2float_rn((rgb.ptr (y)[x]).z) * br_f );
          color = make_uchar3 (r,g,b);
        }
        dst.ptr (y)[x] = color;
      }
    };
    
    __global__ void
    generateImageRGBKernel (const ImageGeneratorRGB ig) {
      ig ();
    }

    void
    generateImage (const MapArr& vmap, const MapArr& nmap, const LightSource& light, 
                                PtrStepSz<uchar3> dst)
    {
      ImageGenerator ig;
      ig.vmap = vmap;
      ig.nmap = nmap;
      ig.light = light;
      ig.dst = dst;

      dim3 block (ImageGenerator::CTA_SIZE_X, ImageGenerator::CTA_SIZE_Y);
      dim3 grid (divUp (dst.cols, block.x), divUp (dst.rows, block.y));

      generateImageKernel<<<grid, block>>>(ig);
      cudaSafeCall (hipGetLastError ());
      cudaSafeCall (hipStreamSynchronize(0));
    }  
    
    
    void
    generateImageRGB (const MapArr& vmap, const MapArr& nmap, 
                      const PtrStepSz<uchar3>& rgb,
                      const LightSource& light, PtrStepSz<uchar3> dst)
    {
      ImageGeneratorRGB ig;
      ig.vmap = vmap;
      ig.nmap = nmap;
      ig.rgb = rgb;
      ig.light = light;
      ig.dst = dst;

      dim3 block (ImageGenerator::CTA_SIZE_X, ImageGenerator::CTA_SIZE_Y);
      dim3 grid (divUp (dst.cols, block.x), divUp (dst.rows, block.y));

      generateImageRGBKernel<<<grid, block>>>(ig);
      cudaSafeCall (hipGetLastError ());
      cudaSafeCall (hipStreamSynchronize(0));
    }  
        
  }
}



